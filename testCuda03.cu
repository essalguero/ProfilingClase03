
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>



__global__ void prueba1(int* datos)
{
	int gid = (blockIdx.x * blockDim.x) + threadIdx.x;
	datos[gid] = gid;
}


int main()
{
	int numbloques = 128;
	int numthreads = 128;

	int* d_datos;
	int* d_datos2;

	hipMalloc((void **) &d_datos, (sizeof(int) * numbloques * (numthreads - 1) - sizeof(int)));
	hipMalloc((void **) &d_datos2, (sizeof(int) * numbloques * (numthreads - 1) - sizeof(int)));

	prueba1<<<numbloques, numthreads>>>(d_datos);

	hipDeviceSynchronize();

	return 0;
}
